#include "hip/hip_runtime.h"
#include "ascii.hpp"

__constant__ const size_t device_size = 71;
__constant__ const uchar device_chars[71] = " .'`^\",:;Il!i><~+_-?][}{1)(|\\/tfjrxnuvczXYUJCLQ0OZmwqpdbkhao*#MW&8%B@$";
__constant__ const float device_divider = 255.0f / static_cast<float>(device_size - 1);

/**
 * Fonction permettant d'obtenir le caractère ASCII correspondant à l'intensité donnée
 * @param intensity L'intensité du caractère à convertie
 * @return Le caractère ASCII correspondant
 */
 __device__
uchar device_convert_intensity(uchar intensity) {
    // Convertion de l'intensité en indice dans le set de caractère
    int rounded = static_cast<int>(static_cast<float>(intensity) / device_divider);
    // Vérification que l'indice n'est pas OOB
    assert(rounded < device_size);
    // Retourne le caractère correspondant
    return device_chars[rounded];
}

/**
 * Fonction de transformation de l'image d'entrée en ASCII
 * @param image L'image source
 * @param output Le stream vers le fichier de sortie
 */
__global__
void asciify(const uchar* data, uchar* candidate, size_t rows, size_t cols) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    uint j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i < cols && j < rows)
        candidate[j * cols + i] = device_convert_intensity(data[j * cols + i]);
}

int main(int argc, char** argv) {

    // Chemin vers le fichier source manquant
    if (argc != 2) missing_argument();

    // Récupération de l'image
    cv::Mat image = cv::imread(argv[1], cv::IMREAD_GRAYSCALE);
    size_t data_size = image.rows * image.cols;

    // Image vide
    if (image.empty() || !image.data) missing_data();

    auto* output_data = new uchar[data_size];
    for (size_t i = 0; i < data_size; i++) output_data[i] = 255;

    uchar* grayscaled;
    uchar* asciified;
    hipError_t e0 = hipMalloc(&grayscaled, data_size);
    if (e0 != hipSuccess) std::cerr << "Error 0 : " << hipGetErrorString(e0) << std::endl;
    hipError_t e1 = hipMalloc(&asciified, data_size);
    if (e1 != hipSuccess) std::cerr << "Error 1 : " << hipGetErrorString(e1) << std::endl;

    hipError_t e2 = hipMemcpy(grayscaled, image.data, data_size, hipMemcpyHostToDevice);
    if (e2 != hipSuccess) std::cerr << "Error 2 : " << hipGetErrorString(e2) << std::endl;

    // TIMERS
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    dim3 thread_size( 32, 4 ); //128 threads
    dim3 block_size( (( image.cols - 1) / (thread_size.x - 2) + 1), (( image.rows - 1 ) / (thread_size.y - 2) + 1) );
    asciify<<<block_size, thread_size, thread_size.x * thread_size.y>>>(grayscaled, asciified, image.rows, image.cols);

    // TIMER
    hipEventRecord(stop);
    hipEventSynchronize( stop );
    float duration;
    hipEventElapsedTime( &duration, start, stop );
    std::cout << "Processing took: " << duration << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipError_t e3 = hipMemcpy(output_data, asciified, data_size, hipMemcpyDeviceToHost);
    if (e3 != hipSuccess) std::cerr << "Error 3 : " << hipGetErrorString(e3) << std::endl;

    // Ouverture du stream vers le fichier de sortie
    std::ofstream output("ascii_gpu.txt");
    size_t i, j;
    for (i = 0; i < image.rows; i++) {
        for (j = 0; j < image.cols; j++) {
            output << output_data[i * image.cols + j] << output_data[i * image.cols + j];
            char x = convert_intensity(image.at<uchar>(i, j));
            if (x != output_data[i * image.cols + j]) {
                std::clog << "Device: " << output_data[i * image.cols + j] << "  ;  Expected: " << x << std::endl;
            }
        }
        output << std::endl;
    }
    output.close();

    // Free les pointers
    delete[] output_data;

    hipFree(grayscaled);
    hipFree(asciified);

    return EXIT_SUCCESS;
}