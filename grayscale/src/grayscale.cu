#include "hip/hip_runtime.h"
#include "grayscale.hpp"

/**
 * Fonction pour convertir un pixel 3 channels en pixel grayscale 1 channel
 * @param pixel Le pixel à convertir
 * @return Le pixel convertie en grayscale
 */
uchar convert_intensity(cv::Vec3b pixel) {
    return (((113 * pixel[0]) + (604 * pixel[1]) + (307 * pixel[2])) / 1024);
}

/**
 * Fonction appliquant l'effet grayscale sur l'image
 * @param image L'image source
 * @return Un pointer vers la nouvelle image
 */
__global__
void grayscale(const uchar* data, uchar* candidate, size_t rows, size_t cols) {

    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    uint j = blockIdx.y * blockDim.y + threadIdx.y;
    if( i < cols && j < rows )
        candidate[ j * cols + i ] = (307 * data[3 * (j * cols + i)] + 604 * data[3 * (j * cols + i ) + 1] + 113 * data[3 * ( j * cols + i ) + 2]) >> 10; // >>10 <=> division par 1024
}

int main(int argc, char** argv) {

    // Chemin vers le fichier source manquant
    if (argc != 2) missing_argument();

    // Récupération de l'image
    cv::Mat image = cv::imread(argv[1], cv::IMREAD_COLOR);
    size_t data_size = image.rows * image.cols * 3;

    // Image vide
    if (image.empty() || !image.data) missing_data();

    auto* output_data = new uchar[data_size];

    uchar* rgb;
    uchar* grayscaled;
    hipError_t e0 = hipMalloc(&rgb, data_size);
    if (e0 != hipSuccess) std::cerr << "Error 0 : " << hipGetErrorString(e0) << std::endl;
    hipError_t e1 = hipMalloc(&grayscaled, data_size);
    if (e1 != hipSuccess) std::cerr << "Error 1 : " << hipGetErrorString(e1) << std::endl;

    hipError_t e2 = hipMemcpy(rgb, image.data, data_size, hipMemcpyHostToDevice);
    if (e2 != hipSuccess) std::cerr << "Error 2 : " << hipGetErrorString(e2) << std::endl;

    // TIMERS
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    dim3 thread_size( 32, 4 ); //128 threads
    dim3 block_size( (( image.cols - 1) / (thread_size.x - 2) + 1), (( image.rows - 1 ) / (thread_size.y - 2) + 1) );
    grayscale<<<block_size, thread_size, thread_size.x * thread_size.y>>>(rgb, grayscaled, image.rows, image.cols);

    // TIMER
    hipEventRecord(stop);
    hipEventSynchronize( stop );
    float duration;
    hipEventElapsedTime( &duration, start, stop );
    std::cout << "time=" << duration << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipError_t e3 = hipMemcpy(output_data, grayscaled, data_size, hipMemcpyDeviceToHost);
    if (e3 != hipSuccess) std::cerr << "Error 3 : " << hipGetErrorString(e3) << std::endl;
    auto* result = new cv::Mat(image.rows, image.cols, CV_8UC1, output_data);
    cv::imwrite("grayscale_gpu.png", *result);

    // Free les pointers
    delete[] output_data;
    delete result;

    hipFree(rgb);
    hipFree(grayscaled);

    return EXIT_SUCCESS;
}
